#include "utils.h"
#include <stdlib.h>

// Implementation of utility functions

// Implementation of CUDA event timer functions
void create_timer(hipEvent_t* start, hipEvent_t* stop) {
    hipEventCreate(start);
    hipEventCreate(stop);
}

void start_timer(hipEvent_t start) {
    hipEventRecord(start);
}

float stop_timer(hipEvent_t start, hipEvent_t stop) {
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    return milliseconds / 1000.0f; // Convert to seconds
}

// Allocate memory for a matrix
double* allocateMatrix(int rows, int cols) {
    if (VERBOSE) printf("Allocating flattened matrix of size %d x %d\n", rows, cols);
    
    // Allocate single contiguous block
    double* mat = (double*)malloc(rows * cols * sizeof(double));
    if (!mat) {
        if (VERBOSE) printf("Failed to allocate matrix\n");
        exit(1);
    }
    
    if (VERBOSE) printf("Flattened matrix allocation successful\n");
    return mat;
}

// Free allocated matrix memory
void freeMatrix(double* mat, int rows) {
    if (VERBOSE) printf("Freeing matrix with %d rows\n", rows);
    free(mat);
    if (VERBOSE) printf("Matrix freed successfully\n");
}

