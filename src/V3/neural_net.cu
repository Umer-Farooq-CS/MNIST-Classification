#include "hip/hip_runtime.h"
#include "neural_net.h"
#include "utils.h"
#include <stdlib.h>
#include <time.h>
#define BLOCK_SIZE 256

NeuralNetwork* createNetwork() {
    if (VERBOSE) printf("Creating neural network...\n");
    NeuralNetwork* net = (NeuralNetwork*)malloc(sizeof(NeuralNetwork));
    if (!net) {
        if (VERBOSE) printf("Failed to allocate neural network\n");
        exit(1);
    }
    
    // Allocate flattened matrices
    net->W1 = (double*)malloc(HIDDEN_SIZE * INPUT_SIZE * sizeof(double));
    net->W2 = (double*)malloc(OUTPUT_SIZE * HIDDEN_SIZE * sizeof(double));
    net->b1 = (double*)calloc(HIDDEN_SIZE, sizeof(double));
    net->b2 = (double*)calloc(OUTPUT_SIZE, sizeof(double));

    if (!net->W1 || !net->W2 || !net->b1 || !net->b2) {
        if (VERBOSE) printf("Failed to allocate weights/biases\n");
        exit(1);
    }

    srand(time(NULL));
    if (VERBOSE) printf("Initializing weights...\n");
    
    // Initialize W1 (flattened)
    for (int i = 0; i < HIDDEN_SIZE; i++) {
        for (int j = 0; j < INPUT_SIZE; j++) {
            net->W1[i * INPUT_SIZE + j] = ((double)rand() / RAND_MAX) * 0.01;
        }
    }

    // Initialize W2 (flattened)
    for (int i = 0; i < OUTPUT_SIZE; i++) {
        for (int j = 0; j < HIDDEN_SIZE; j++) {
            net->W2[i * HIDDEN_SIZE + j] = ((double)rand() / RAND_MAX) * 0.01;
        }
    }

    // Allocate and copy device memory
    checkCudaError(hipMalloc(&net->d_W1, HIDDEN_SIZE * INPUT_SIZE * sizeof(double)), "hipMalloc d_W1");
    checkCudaError(hipMalloc(&net->d_W2, OUTPUT_SIZE * HIDDEN_SIZE * sizeof(double)), "hipMalloc d_W2");
    checkCudaError(hipMalloc(&net->d_b1, HIDDEN_SIZE * sizeof(double)), "hipMalloc d_b1");
    checkCudaError(hipMalloc(&net->d_b2, OUTPUT_SIZE * sizeof(double)), "hipMalloc d_b2");
    checkCudaError(hipMalloc(&net->d_input, INPUT_SIZE * sizeof(double)), "hipMalloc d_input");
    checkCudaError(hipMalloc(&net->d_hidden, HIDDEN_SIZE * sizeof(double)), "hipMalloc d_hidden");
    checkCudaError(hipMalloc(&net->d_output, OUTPUT_SIZE * sizeof(double)), "hipMalloc d_output");

    // Copy initial values to device
    checkCudaError(hipMemcpy(net->d_W1, net->W1, HIDDEN_SIZE * INPUT_SIZE * sizeof(double), hipMemcpyHostToDevice), "hipMemcpy d_W1");
    checkCudaError(hipMemcpy(net->d_W2, net->W2, OUTPUT_SIZE * HIDDEN_SIZE * sizeof(double), hipMemcpyHostToDevice), "hipMemcpy d_W2");
    checkCudaError(hipMemcpy(net->d_b1, net->b1, HIDDEN_SIZE * sizeof(double), hipMemcpyHostToDevice), "hipMemcpy d_b1");
    checkCudaError(hipMemcpy(net->d_b2, net->b2, OUTPUT_SIZE * sizeof(double), hipMemcpyHostToDevice), "hipMemcpy d_b2");

    if (VERBOSE) {
        printf("Weight initialization complete\n");
        printf("W1[0][0]: %.6f\n", net->W1[0]);  // First element of flattened W1
        printf("W2[0][0]: %.6f\n", net->W2[0]);  // First element of flattened W2
        printf("b1[0]: %.6f\n", net->b1[0]);
        printf("b2[0]: %.6f\n", net->b2[0]);
    }
    
    if (VERBOSE) printf("Neural network created successfully\n");
    return net;
}

// ReLU activation kernel
__global__ void relu_kernel(double* x, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        x[idx] = (x[idx] > 0) ? x[idx] : 0;
    }
}

// Softmax activation kernel
__global__ void softmax_kernel(double* x, int size) {
    __shared__ double sum;
    double val = 0;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < size) {
        val = exp(x[idx]);
    }
    
    // Thread 0 in block initializes sum
    if (threadIdx.x == 0) sum = 0;
    __syncthreads();
    
    // Atomic add to sum
    atomicAdd(&sum, val);
    __syncthreads();
    
    if (idx < size) {
        x[idx] = val / sum;
    }
}

void backward(NeuralNetwork* net, double* d_input, double* d_target) {
    if (VERBOSE) printf("\nStarting GPU backward pass...\n");

    int blockSize = 256;
    int numBlocksOutput = (OUTPUT_SIZE + blockSize - 1) / blockSize;
    int numBlocksHidden = (HIDDEN_SIZE + blockSize - 1) / blockSize;
    int numBlocksW2 = ((OUTPUT_SIZE * HIDDEN_SIZE) + blockSize - 1) / blockSize;
    int numBlocksW1 = ((HIDDEN_SIZE * INPUT_SIZE) + blockSize - 1) / blockSize;
    
    // Step 1: Compute output gradients: d_output = d_output - d_target.
    computeDOutputKernel<<<numBlocksOutput, blockSize>>>(net->d_output, d_target, OUTPUT_SIZE);
    checkCudaError(hipGetLastError(), "Kernel launch: computeDOutputKernel");
    checkCudaError(hipDeviceSynchronize(), "hipDeviceSynchronize after computeDOutputKernel");
    
    // Step 2: Save the forward hidden activations.
    double* d_hidden_forward;
    checkCudaError(hipMalloc(&d_hidden_forward, HIDDEN_SIZE * sizeof(double)), "hipMalloc d_hidden_forward");
    checkCudaError(hipMemcpy(d_hidden_forward, net->d_hidden, HIDDEN_SIZE * sizeof(double), hipMemcpyDeviceToDevice), "hipMemcpy d_hidden_forward");
    
    // Allocate a temporary device array for the hidden gradients.
    double* d_hidden_grad;
    checkCudaError(hipMalloc(&d_hidden_grad, HIDDEN_SIZE * sizeof(double)), "hipMalloc d_hidden_grad");
    
    // Step 3: Compute hidden layer gradients.
    computeDHiddenKernel<<<numBlocksHidden, blockSize>>>(net->d_W2, net->d_output, d_hidden_forward, d_hidden_grad, HIDDEN_SIZE, OUTPUT_SIZE);
    checkCudaError(hipGetLastError(), "Kernel launch: computeDHiddenKernel");
    checkCudaError(hipDeviceSynchronize(), "hipDeviceSynchronize after computeDHiddenKernel");
    
    // Step 4: Update W2 using the forward hidden activations.
    updateW2Kernel<<<numBlocksW2, blockSize>>>(net->d_W2, net->d_output, d_hidden_forward, HIDDEN_SIZE, OUTPUT_SIZE, LEARNING_RATE);
    checkCudaError(hipGetLastError(), "Kernel launch: updateW2Kernel");
    checkCudaError(hipDeviceSynchronize(), "hipDeviceSynchronize after updateW2Kernel");
    
    // Step 5: Update W1 using the computed hidden gradients.
    updateW1Kernel<<<numBlocksW1, blockSize>>>(net->d_W1, d_hidden_grad, d_input, INPUT_SIZE, HIDDEN_SIZE, LEARNING_RATE);
    checkCudaError(hipGetLastError(), "Kernel launch: updateW1Kernel");
    checkCudaError(hipDeviceSynchronize(), "hipDeviceSynchronize after updateW1Kernel");
    
    // Step 6: Update biases.
    updateBiasesKernel<<<numBlocksOutput, blockSize>>>(net->d_b2, net->d_output, OUTPUT_SIZE, LEARNING_RATE);
    checkCudaError(hipGetLastError(), "Kernel launch: updateBiasesKernel (d_b2)");
    checkCudaError(hipDeviceSynchronize(), "hipDeviceSynchronize after updateBiasesKernel (d_b2)");
    
    updateBiasesKernel<<<numBlocksHidden, blockSize>>>(net->d_b1, d_hidden_grad, HIDDEN_SIZE, LEARNING_RATE);
    checkCudaError(hipGetLastError(), "Kernel launch: updateBiasesKernel (d_b1)");
    checkCudaError(hipDeviceSynchronize(), "hipDeviceSynchronize after updateBiasesKernel (d_b1)");
    
    // Free temporary arrays.
    checkCudaError(hipFree(d_hidden_forward), "hipFree d_hidden_forward");
    checkCudaError(hipFree(d_hidden_grad), "hipFree d_hidden_grad");

    if (VERBOSE) printf("GPU backward pass completed\n");
}


// Backpropagation on host
void backward(NeuralNetwork* net, double* input, double* hidden, double* output, double* target) {
    if (VERBOSE) printf("\nStarting backward pass...\n");
    double d_output[OUTPUT_SIZE], d_hidden[HIDDEN_SIZE];

    // Compute output layer gradient.
    if (VERBOSE) printf("Computing output gradients...\n");
    for (int i = 0; i < OUTPUT_SIZE; i++)
        d_output[i] = output[i] - target[i];
    
    if (VERBOSE) {
        printf("Output gradients: ");
        for (int i = 0; i < OUTPUT_SIZE; i++) printf("%.4f ", d_output[i]);
        printf("\n");
    }

    // Compute hidden layer gradient.
    if (VERBOSE) printf("Computing hidden gradients...\n");
    for (int i = 0; i < HIDDEN_SIZE; i++) {
        d_hidden[i] = 0;
        for (int j = 0; j < OUTPUT_SIZE; j++)
            d_hidden[i] += net->W2[j * HIDDEN_SIZE + i] * d_output[j];
        d_hidden[i] *= (hidden[i] > 0);
    }
    
    if (VERBOSE) {
        printf("Hidden gradients (first 5): ");
        for (int i = 0; i < 5; i++) printf("%.4f ", d_hidden[i]);
        printf("\n");
    }

    // Update weights (gradient descent).
    if (VERBOSE) printf("Updating weights...\n");
    for (int i = 0; i < OUTPUT_SIZE; i++) {
        for (int j = 0; j < HIDDEN_SIZE; j++) {
            net->W2[i * HIDDEN_SIZE + j] -= LEARNING_RATE * d_output[i] * hidden[j];
        }
    }

    for (int i = 0; i < HIDDEN_SIZE; i++) {
        for (int j = 0; j < INPUT_SIZE; j++) {
            net->W1[i * INPUT_SIZE + j] -= LEARNING_RATE * d_hidden[i] * input[j];
        }
    }

    // Update biases.
    if (VERBOSE) printf("Updating biases...\n");
    for (int i = 0; i < OUTPUT_SIZE; i++)
        net->b2[i] -= LEARNING_RATE * d_output[i];

    for (int i = 0; i < HIDDEN_SIZE; i++)
        net->b1[i] -= LEARNING_RATE * d_hidden[i];
    
    // Update device weights.
    checkCudaError(hipMemcpy(net->d_W1, net->W1, HIDDEN_SIZE * INPUT_SIZE * sizeof(double), hipMemcpyHostToDevice), "hipMemcpy update d_W1");
    checkCudaError(hipMemcpy(net->d_W2, net->W2, OUTPUT_SIZE * HIDDEN_SIZE * sizeof(double), hipMemcpyHostToDevice), "hipMemcpy update d_W2");
    checkCudaError(hipMemcpy(net->d_b1, net->b1, HIDDEN_SIZE * sizeof(double), hipMemcpyHostToDevice), "hipMemcpy update d_b1");
    checkCudaError(hipMemcpy(net->d_b2, net->b2, OUTPUT_SIZE * sizeof(double), hipMemcpyHostToDevice), "hipMemcpy update d_b2");
    
    if (VERBOSE) {
        printf("Updated W2[0][0]: %.6f\n", net->W2[0]);
        printf("Updated W1[0][0]: %.6f\n", net->W1[0]);
        printf("Updated b2[0]: %.6f\n", net->b2[0]);
        printf("Updated b1[0]: %.6f\n", net->b1[0]);
        printf("Backward pass completed\n");
    }
}

// ----------------------------------------------------------------------------
// Optimized kernel for matrix–vector multiplication using shared memory reduction.
// Each block handles one output row. Each thread in the block computes a partial sum 
// over a subset of columns.
__global__ void matrixVectorMultiplySM(const double* __restrict__ W, 
                                         const double* __restrict__ x, 
                                         const double* __restrict__ b, 
                                         double* __restrict__ y, 
                                         int rows, int cols) {
    extern __shared__ double sdata[];
    int row = blockIdx.x;  // each block does one row.
    int tid = threadIdx.x;
    double sum = 0.0;
    // Each thread sums over columns strided by blockDim.x.
    for (int j = tid; j < cols; j += blockDim.x) {
        sum += W[row * cols + j] * x[j];
    }
    sdata[tid] = sum;
    __syncthreads();
    // Perform parallel reduction in shared memory.
    for (unsigned int s = blockDim.x/2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    if (tid == 0) {
        // Add the corresponding bias and write out result.
        y[row] = sdata[0] + b[row];
    }
}

// ----------------------------------------------------------------------------
// Optimized softmax kernel using shared memory reduction for the sum.
// This version first computes the maximum value (for numerical stability),
// then computes exponentials and reduces them.
__global__ void softmaxKernelOpt(double* x, int size) {
    extern __shared__ double sdata[];
    int tid = threadIdx.x;
    // Use the first block (assume one block launched for the vector)
    // First, find max value using parallel reduction.
    double max_val = -1e20;
    for (int i = tid; i < size; i += blockDim.x) {
        double tmp = x[i];
        if (tmp > max_val) max_val = tmp;
    }
    sdata[tid] = max_val;
    __syncthreads();
    // Reduction to get global max.
    for (unsigned int s = blockDim.x/2; s > 0; s >>= 1) {
        if (tid < s)
            sdata[tid] = max(sdata[tid], sdata[tid + s]);
        __syncthreads();
    }
    max_val = sdata[0];
    // Now compute exponentials (using max_val for stability)
    double sum = 0.0;
    for (int i = tid; i < size; i += blockDim.x) {
        double exp_val = exp(x[i] - max_val);
        x[i] = exp_val;  // store exponentials temporarily
        sum += exp_val;
    }
    sdata[tid] = sum;
    __syncthreads();
    // Reduce to get the total sum.
    for (unsigned int s = blockDim.x/2; s > 0; s >>= 1) {
        if (tid < s)
            sdata[tid] += sdata[tid+s];
        __syncthreads();
    }
    double total = sdata[0];
    // Finally normalize.
    for (int i = tid; i < size; i += blockDim.x) {
        x[i] /= total;
    }
}

// ----------------------------------------------------------------------------
// Forward pass function rewritten to use asynchronous copies and a stream.
void forward(NeuralNetwork* net, double* input, double* hidden, double* output) {
    if (VERBOSE) printf("\nStarting forward pass...\n");
    
    hipStream_t stream;
    checkCudaError(hipStreamCreate(&stream), "hipStreamCreate");

    // Asynchronously copy input to device.
    checkCudaError(hipMemcpyAsync(net->d_input, input, INPUT_SIZE * sizeof(double), hipMemcpyHostToDevice, stream),
                   "hipMemcpyAsync input");

    int blockSize = BLOCK_SIZE;
    int numBlocksHidden = HIDDEN_SIZE;  // one block per row in hidden layer.
    size_t sharedSize = blockSize * sizeof(double);
    
    // Launch optimized matrix-vector multiplication for hidden layer.
    matrixVectorMultiplySM<<<numBlocksHidden, blockSize, sharedSize, stream>>>(
        net->d_W1, net->d_input, net->d_b1, net->d_hidden, HIDDEN_SIZE, INPUT_SIZE);
    checkCudaError(hipGetLastError(), "Kernel launch: matrixVectorMultiplySM (hidden)");

    // Launch ReLU kernel (we assume it is memory‐bound so shared memory is not needed here).
    int numBlocks = (HIDDEN_SIZE + blockSize - 1) / blockSize;
    relu_kernel<<<numBlocks, blockSize, 0, stream>>>(net->d_hidden, HIDDEN_SIZE);
    checkCudaError(hipGetLastError(), "Kernel launch: relu_kernel");

    // For the output layer, use the same optimized kernel.
    int numBlocksOutput = OUTPUT_SIZE;  // one block per output neuron.
    matrixVectorMultiplySM<<<numBlocksOutput, blockSize, sharedSize, stream>>>(
        net->d_W2, net->d_hidden, net->d_b2, net->d_output, OUTPUT_SIZE, HIDDEN_SIZE);
    checkCudaError(hipGetLastError(), "Kernel launch: matrixVectorMultiplySM (output)");

    // Launch optimized softmax kernel.
    // Assume one block is sufficient if OUTPUT_SIZE is small.
    softmaxKernelOpt<<<1, BLOCK_SIZE, BLOCK_SIZE * sizeof(double), stream>>>(net->d_output, OUTPUT_SIZE);
    checkCudaError(hipGetLastError(), "Kernel launch: softmaxKernelOpt");

    // Copy results back to host asynchronously.
    checkCudaError(hipMemcpyAsync(hidden, net->d_hidden, HIDDEN_SIZE * sizeof(double), hipMemcpyDeviceToHost, stream),
                   "hipMemcpyAsync hidden");
    checkCudaError(hipMemcpyAsync(output, net->d_output, OUTPUT_SIZE * sizeof(double), hipMemcpyDeviceToHost, stream),
                   "hipMemcpyAsync output");

    // Wait for stream to complete.
    checkCudaError(hipStreamSynchronize(stream), "hipStreamSynchronize forward");
    checkCudaError(hipStreamDestroy(stream), "hipStreamDestroy forward");

    if (VERBOSE) {
        printf("Post-ReLU hidden (first 5): ");
        for (int i = 0; i < 5; i++) printf("%.4f ", hidden[i]);
        printf("\n");
        printf("Post-softmax output: ");
        for (int i = 0; i < OUTPUT_SIZE; i++) printf("%.4f ", output[i]);
        printf("\n");
        printf("Forward pass completed\n");
    }
}

// Kernel to compute d_output = d_output - d_target, for the output layer gradients.
__global__ void computeDOutputKernel(double* d_output, const double* d_target, int outputSize) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < outputSize) {
        d_output[i] = d_output[i] - d_target[i];
    }
}

// Kernel to compute d_hidden_grad for each hidden neuron.
// Uses the layer-2 weights and the computed d_output.
// The forward activation (before replacing with gradients) is in d_hidden_forward.
__global__ void computeDHiddenKernel(const double* d_W2, const double* d_output, 
                                       const double* d_hidden_forward, double* d_hidden_grad,
                                       int hiddenSize, int outputSize) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < hiddenSize) {
        double sum = 0.0;
        for (int j = 0; j < outputSize; j++) {
            sum += d_W2[j * hiddenSize + i] * d_output[j];
        }
        // ReLU derivative: if forward hidden activation > 0, derivative is 1 else 0.
        d_hidden_grad[i] = (d_hidden_forward[i] > 0.0) ? sum : 0.0;
    }
}

// Kernel to update the weights for the output layer (W2).
// Uses the computed output gradients and the forward hidden activations.
__global__ void updateW2Kernel(double* d_W2, const double* d_output, 
                               const double* d_hidden_forward, int hiddenSize, 
                               int outputSize, double learning_rate) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = outputSize * hiddenSize;
    if (idx < total) {
        int out_idx = idx / hiddenSize;  // index for output neuron
        int hid_idx = idx % hiddenSize;    // index for hidden neuron
        d_W2[idx] -= learning_rate * d_output[out_idx] * d_hidden_forward[hid_idx];
    }
}

// Kernel to update the weights for the hidden layer (W1).
// Uses the computed hidden gradients and the input.
__global__ void updateW1Kernel(double* d_W1, const double* d_hidden_grad, 
                               const double* d_input, int inputSize, int hiddenSize, 
                               double learning_rate) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = hiddenSize * inputSize;
    if (idx < total) {
        int hid_idx = idx / inputSize;
        int in_idx  = idx % inputSize;
        d_W1[idx] -= learning_rate * d_hidden_grad[hid_idx] * d_input[in_idx];
    }
}

// Kernel to update biases, used for both layers.
__global__ void updateBiasesKernel(double* d_bias, const double* d_grad, int size, double learning_rate) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        d_bias[i] -= learning_rate * d_grad[i];
    }
}


// We now show an example of using asynchronous streams in the training loop.
// Instead of processing one sample at a time strictly sequentially, we use multiple streams.
// In a real system you would batch many samples together.
void train(NeuralNetwork* net, double* images, double* labels, int numImages) {
    if (VERBOSE) printf("\nStarting training...\n");

    hipEvent_t total_start, total_stop;
    create_timer(&total_start, &total_stop);
    start_timer(total_start);

    // Create a stream pool. For demonstration, we use two streams.
    const int numStreams = 2;
    hipStream_t streams[numStreams];
    for (int s = 0; s < numStreams; s++) {
        checkCudaError(hipStreamCreate(&streams[s]), "hipStreamCreate in train");
    }

    for (int epoch = 0; epoch < EPOCHS; epoch++) {
        hipEvent_t epoch_start, epoch_stop;
        create_timer(&epoch_start, &epoch_stop);
        start_timer(epoch_start);

        double loss = 0.0;
        int correct = 0;

        if (VERBOSE) printf("\nEpoch %d/%d\n", epoch+1, EPOCHS);

        // For simplicity, process samples in a loop and alternate streams.
        for (int i = 0; i < numImages; i++) {
            int streamId = i % numStreams;
            
            // Allocate device memory for target and copy asynchronously using the stream.
            double* d_target;
            checkCudaError(hipMalloc(&d_target, OUTPUT_SIZE * sizeof(double)), "hipMalloc d_target in train");
            checkCudaError(hipMemcpyAsync(d_target, &labels[i * OUTPUT_SIZE], OUTPUT_SIZE * sizeof(double), 
                                             hipMemcpyHostToDevice, streams[streamId]), "hipMemcpyAsync d_target in train");

            double hidden[HIDDEN_SIZE], output[OUTPUT_SIZE];
            // Launch forward pass on the chosen stream.
            // (For simplicity, here we call forward which creates its own stream.
            // In a real implementation you would modify forward to accept a stream parameter.)
            forward(net, &images[i * INPUT_SIZE], hidden, output);

            // Call the GPU backward pass (using the stream as appropriate).
            backward(net, net->d_input, d_target);

            // Compute loss & accuracy on the host from the output.
            for (int k = 0; k < OUTPUT_SIZE; k++) {
                loss -= labels[i * OUTPUT_SIZE + k] * log(output[k]);
            }
            int pred = 0, actual = 0;
            for (int j = 0; j < OUTPUT_SIZE; j++) {
                if (output[j] > output[pred]) pred = j;
                if (labels[i * OUTPUT_SIZE + j] > labels[i * OUTPUT_SIZE + actual]) actual = j;
            }
            if (pred == actual) correct++;

            checkCudaError(hipFree(d_target), "hipFree d_target in train");
        }

        printf("Epoch %d - Loss: %.4f - Train Accuracy: %.2f%% - Time: %.3fs\n",
               epoch + 1, loss / numImages, (correct / (double)numImages) * 100, 
               stop_timer(epoch_start, epoch_stop));
    }

    // Destroy streams.
    for (int s = 0; s < numStreams; s++) {
        checkCudaError(hipStreamDestroy(streams[s]), "hipStreamDestroy in train");
    }

    printf("Total training time: %.3fs\n", stop_timer(total_start, total_stop));

    if (VERBOSE) printf("Training completed\n");
}

// ----------------------------------------------------------------------------
// The evaluate and freeNetwork functions remain similar, though you can use 
// asynchronous copies in evaluate if desired.
// ----------------------------------------------------------------------------

void evaluate(NeuralNetwork* net, double* images, double* labels, int numImages) {
    if (VERBOSE) printf("\nStarting evaluation...\n");
    int correct = 0;
    
    for (int i = 0; i < numImages; i++) {
        double hidden[HIDDEN_SIZE], output[OUTPUT_SIZE];
        forward(net, &images[i * INPUT_SIZE], hidden, output);
        
        int pred = 0, actual = 0;
        for (int j = 0; j < OUTPUT_SIZE; j++) {
            if (output[j] > output[pred]) pred = j;
            if (labels[i * OUTPUT_SIZE + j] > labels[i * OUTPUT_SIZE + actual]) actual = j;
        }
        if (pred == actual) correct++;
        
        if (VERBOSE && i < 3) {
            printf("Sample %d - Predicted: %d, Actual: %d\n", i, pred, actual);
            printf("Output probabilities: ");
            for (int j = 0; j < OUTPUT_SIZE; j++) printf("%.2f ", output[j]);
            printf("\n");
        }
    }
    
    printf("Test Accuracy: %.2f%%\n", (correct / (double)numImages) * 100);
    if (VERBOSE) printf("Evaluation completed\n");
}


void freeNetwork(NeuralNetwork* net) {
    if (VERBOSE) printf("Freeing neural network...\n");
    
    // Free host memory
    free(net->W1);
    free(net->W2);
    free(net->b1);
    free(net->b2);
    
    // Free device memory
    checkCudaError(hipFree(net->d_W1), "hipFree d_W1");
    checkCudaError(hipFree(net->d_W2), "hipFree d_W2");
    checkCudaError(hipFree(net->d_b1), "hipFree d_b1");
    checkCudaError(hipFree(net->d_b2), "hipFree d_b2");
    checkCudaError(hipFree(net->d_input), "hipFree d_input");
    checkCudaError(hipFree(net->d_hidden), "hipFree d_hidden");
    checkCudaError(hipFree(net->d_output), "hipFree d_output");
    
    free(net);
    if (VERBOSE) printf("Neural network freed\n");
}
